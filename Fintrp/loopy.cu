#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>




extern __shared__ double cache[];

__global__ void kernel(int *Ss, int *Nn, int *mask, double *xyz,
		       double *cost){

  int i1 = blockIdx.x;
  int i2 = threadIdx.x;
  int j = threadIdx.y;
  int dimx= blockDim.x, dimy = blockDim.y;
  int cacheIndex = j*dimx+i2, cIndexMax = dimx*dimy;


  cache[cacheIndex] = 0.5;

  __syncthreads();

  //Somar todos os indices do cache aqui
  //ofset separado

  int i;
  for(i=1; i<cIndexMax; i++)
    cache[0]+=cache[i];
  
  cost[i1] += cache[0];
 
}

int main()
{
  //Initializing
  int N = 3, S = 10 ;
  double xyz[3][N][S], linxyz[3*N*S],soma;
  double cost[S];
  int mask[3][N]={0}, linmask[3*N];
  long int i1,i2;
  long int j=0,k=0;

  //mask
    for(k=0; k<3; k++){
      for(j=0; j<N; j++){
	mask[k][j] = 1;
	if(j%(k+1)==0)
	  mask[k][j] = 1;	
      }
    }
   
    for(k=0; k<3; k++){
      for(j=0; j<N; j++){
	linmask[j+N*k] = mask[k][j];
      }
    }
    //mask

    for(k=0; k<3; k++){
      for(j=0; j<N;j++){
	for(i1=0; i1<S; i1++){
	  xyz[k][j][i1] = 0.1*(i1+S*j + S*N*k);
	  linxyz[i1+S*j + S*N*k] = xyz[k][j][i1];
	}
      }
    }


    //CPU
    soma = 0;

  for (i1 = 0  ; i1 < S ; ++i1) {    
    for (i2 = 0;   i2< S ; ++i2) {
      if(i1!=i2){
	soma = 0;
	for(j=0;   j<N;   j++){
	  
	  for(k=0;   k<3;    k++){

	    if( linmask [N*k+j] ){
	      soma+=(linxyz[k*N*S+j*S+i1] - linxyz[k*N*S+j*S+i2])*(linxyz[k*N*S+j*S+i1] - linxyz[k*N*S+j*S+i2]);
	    }
	    
	  } //for k
	} //for j
       	cost[i1]+=soma;
      } //for if
    } //for i2
  } //for i1

  //GPU

  int *devN, *devS;
  
  hipMalloc((void **)&devN, sizeof(int));
  hipMalloc((void **)&devS, sizeof(int));

  hipMemcpy(devN, &N, sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(devS, &S, sizeof(int), hipMemcpyHostToDevice);

  int *dmask;
  hipMalloc((void **)&dmask, sizeof(linmask));

  hipMemcpy(dmask, linmask, sizeof(linmask), hipMemcpyHostToDevice);

  double *d_xyz, *d_cost, cost2[S];
  hipMalloc((void **)&d_xyz, sizeof(linxyz));
  hipMalloc((void **)&d_cost, S*sizeof(double));

  hipMemcpy(d_xyz, linxyz, sizeof(linxyz), hipMemcpyHostToDevice);

  dim3    grids(S);
  dim3    threads(S,N);

 
  
  kernel<<<grids,threads,S*N*sizeof(double)>>>(devS, devN, dmask, d_xyz, d_cost);

  hipMemcpy(cost2, d_cost, S*sizeof(double), hipMemcpyDeviceToHost);

  printf("%f %f \n", cost2[0], cost2[1]);

  hipFree(devN); hipFree(devS); hipFree(dmask);
  hipFree(d_xyz); hipFree(d_cost);
  
			    /*__global__ void kernel
(int *Ss, int *Nn, int *mask, double *xyz,  double *cost)*/
  return 0;
}
